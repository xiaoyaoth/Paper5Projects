#include "hip/hip_runtime.h"
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""


#define N 10000


__global__ void kernel_a()
{
	float sum = 0.0;
	for (int i = 0; i < N; i++)
	{
		sum = sum + cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1);
	}
}

__global__ void kernel_b()
{
	float sum = 0.0;
	for (int i = 0; i < N / 2; i++)
	{
		sum = sum + cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1);
	}
}

__global__ void kernel_c()
{
	float sum = 0.0;
	for (int i = 0; i < N / 4; i++)
	{
		sum = sum + cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1);
	}
}

__global__ void kernel_d()
{
	float sum = 0.0;
	for (int i = 0; i < N / 8; i++)
	{
		sum = sum + cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1)*cos(0.1);
	}
}

int main()
{

	int n_streams = 8;
	int n_threads = 4;
	hipError_t cudaStatus;

	// Allocate and initialize an array of stream handles
	hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
	for (int i = 0; i < n_streams; i++)
	{
		hipStreamCreate(&(streams[i]));
	}

	// Parallel section
	for (int iter = 0; iter < 10; iter++) {
#pragma omp parallel for num_threads(n_threads)

		for (int i = 0; i < n_streams; i++)
		{

			kernel_a << <1, 1, 0, streams[i] >> >();

			kernel_b << <1, 1, 0, streams[i] >> >();

			kernel_c << <1, 1, 0, streams[i] >> >();

			kernel_d << <1, 1, 0, streams[i] >> >();
		}
	}
	

	// release all stream
	for (int i = 0; i < n_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
	free(streams);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();

	return 0;
}