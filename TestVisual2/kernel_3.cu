#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <fstream>
#include "SocialForceGPU.h"
#include <omp.h>
__global__ void testFunc() {

}

namespace NeighborModule {
	__device__ int zcode(int x, int y) {
		//return x * NUM_CELL + y;
		x &= 0x0000ffff;					// x = ---- ---- ---- ---- fedc ba98 7654 3210
		y &= 0x0000ffff;					// x = ---- ---- ---- ---- fedc ba98 7654 3210
		x = (x ^ (x << 8)) & 0x00ff00ff; // x = ---- ---- fedc ba98 ---- ---- 7654 3210
		y = (y ^ (y << 8)) & 0x00ff00ff; // x = ---- ---- fedc ba98 ---- ---- 7654 3210
		y = (y ^ (y << 4)) & 0x0f0f0f0f; // x = ---- fedc ---- ba98 ---- 7654 ---- 3210
		x = (x ^ (x << 4)) & 0x0f0f0f0f; // x = ---- fedc ---- ba98 ---- 7654 ---- 3210
		y = (y ^ (y << 2)) & 0x33333333; // x = --fe --dc --ba --98 --76 --54 --32 --10
		x = (x ^ (x << 2)) & 0x33333333; // x = --fe --dc --ba --98 --76 --54 --32 --10
		y = (y ^ (y << 1)) & 0x55555555; // x = -f-e -d-c -b-a -9-8 -7-6 -5-4 -3-2 -1-0
		x = (x ^ (x << 1)) & 0x55555555; // x = -f-e -d-c -b-a -9-8 -7-6 -5-4 -3-2 -1-0
		return x | (y << 1);
	}

	__device__ int zcode(const double2 &loc) {
		int ix = loc.x / (ENV_DIM / NUM_CELL);
		int iy = loc.y / (ENV_DIM / NUM_CELL);
		return zcode(ix, iy);
	}

	__device__ int zcode(SocialForceAgent *agent) {
		return zcode(agent->data.loc);
	}

	__device__ void swap(SocialForceAgent** agentPtrs, int a, int b) {
		SocialForceAgent* temp = agentPtrs[a];
		agentPtrs[a] = agentPtrs[b];
		agentPtrs[b] = temp;
	}

	__device__ void quickSortByAgentLoc(SocialForceAgent** agentPtrs, hiprandState &rState, int l, int r) {
		if (l == r)
			return;
		int pi = l + hiprand(&rState) % (r - l);
		swap(agentPtrs, l, pi);
		SocialForceAgent* pivot = agentPtrs[l];

		int i = l + 1, j = l + 1;
		for (; j < r; j++) {
			if (zcode(agentPtrs[j]) < zcode(pivot)) {
				swap(agentPtrs, i, j);
				i++;
			}
		}
		swap(agentPtrs, l, i - 1);
		quickSortByAgentLoc(agentPtrs, rState, l, i - 1);
		quickSortByAgentLoc(agentPtrs, rState, i, r);
	}

	__global__ void sortAgentByLocKernel(SocialForceAgent** agentPtrsToSort, hiprandState *rState, int numCap) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		hiprandState &rStateLocal = *rState;
		if (idx == 0)
			quickSortByAgentLoc(agentPtrsToSort, rStateLocal, 0, numCap);
	}

	__global__ void setCidStartEndKernel(SocialForceAgent** contextSorted, int* cidStarts, int* cidEnds, int numCap) {
		const int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numCap && idx > 0) {
			int cid = zcode(contextSorted[idx]);
			int cidPrev = zcode(contextSorted[idx - 1]);
			if (cid != cidPrev) {
				cidStarts[cid] = idx;
				cidEnds[cidPrev] = idx;
			}
		}
		if (idx == 0) {
			int cid = zcode(contextSorted[0]);
			cidStarts[cid] = 0;

			cid = zcode(contextSorted[numCap - 1]);
			cidEnds[cid] = numCap;
		}
	}
}

extern "C"
void runTest() {
	testFunc << <32, 32 >> >();
}

/* helper functions and data structures*/
#define checkCudaErrors(err)	__checkCudaErrors(err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
	if (hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString(err));
		exit(-1);
	}
}

namespace APUtil {
	__global__ void hookPointerAndDataKernel(SocialForceAgent** agentPtrArray, SocialForceAgent* agentArray, int numCap) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < numCap) agentPtrArray[index] = &agentArray[index];
	}
};

extern "C"
void hookPointerAndData(SocialForceAgent** agentPtrArray, SocialForceAgent* agentArray, int numCap) {
	int gSize = GRID_SIZE(numCap);
	APUtil::hookPointerAndDataKernel << <gSize, BLOCK_SIZE >> >(agentPtrArray, agentArray, numCap);
}

__device__ double SocialForceAgent::correctCrossBoader(double val, double limit)
{
	if (val >= limit)
		return limit - 0.001;
	else if (val < 0)
		return 0;
	return val;
}
void SocialForceAgent::computeIndivSocialForceRoom(const SocialForceAgentData &myData, const SocialForceAgentData &otherData, double2 &fSum){
	double cMass = 100;
	//my data
	const double2& loc = myData.loc;
	const double2& goal = myData.goal;
	const double2& velo = myData.velocity;
	const double& v0 = myData.v0;
	const double& mass = myData.mass;
	//other's data
	const double2& locOther = otherData.loc;
	const double2& goalOther = otherData.goal;
	const double2& veloOther = otherData.velocity;
	const double& v0Other = otherData.v0;
	const double& massOther = otherData.mass;

	double d = 1e-15 + sqrt((loc.x - locOther.x) * (loc.x - locOther.x) + (loc.y - locOther.y) * (loc.y - locOther.y));
	double dDelta = mass / cMass + massOther / cMass - d;
	double fExp = A * exp(dDelta / B);
	double fKg = dDelta < 0 ? 0 : k1 *dDelta;
	double nijx = (loc.x - locOther.x) / d;
	double nijy = (loc.y - locOther.y) / d;
	double fnijx = (fExp + fKg) * nijx;
	double fnijy = (fExp + fKg) * nijy;
	double fkgx = 0;
	double fkgy = 0;
	if (dDelta > 0) {
		double tix = -nijy;
		double tiy = nijx;
		fkgx = k2 * dDelta;
		fkgy = k2 * dDelta;
		double vijDelta = (veloOther.x - velo.x) * tix + (veloOther.y - velo.y) * tiy;
		fkgx = fkgx * vijDelta * tix;
		fkgy = fkgy * vijDelta * tiy;
	}
	fSum.x += fnijx + fkgx;
	fSum.y += fnijy + fkgy;
}
__device__ void SocialForceAgent::computeForceWithWall(const SocialForceAgentData &dataLocal, obstacleLine &wall, const int &cMass, double2 &fSum) {
	double2 wl = make_double2(wall.ex - wall.sx, wall.ey - wall.sy);
	if (length(wl) == 0) return;
	double diw, crx, cry;
	const double2 &loc = dataLocal.loc;

	diw = wall.pointToLineDist(loc, crx, cry);
	double virDiw = DIST(loc.x, loc.y, crx, cry);

	if (virDiw == 0)
		return;

	double niwx = (loc.x - crx) / virDiw;
	double niwy = (loc.y - cry) / virDiw;
	double drw = dataLocal.mass / cMass - diw;
	double fiw1 = A * exp(drw / B);
	if (drw > 0)
		fiw1 += k1 * drw;
	double fniwx = fiw1 * niwx;
	double fniwy = fiw1 * niwy;

	double fiwKgx = 0, fiwKgy = 0;
	if (drw > 0)
	{
		double fiwKg = k2 * drw * (dataLocal.velocity.x * (-niwy) + dataLocal.velocity.y * niwx);
		fiwKgx = fiwKg * (-niwy);
		fiwKgy = fiwKg * niwx;
	}

	fSum.x += fniwx - fiwKgx;
	fSum.y += fniwy - fiwKgy;
}
__device__ void SocialForceAgent::computeWallImpaction(const SocialForceAgentData &dataLocal, obstacleLine &wall, const double2 &newVelo, const double &tick, double &mint){
	double crx, cry, tt;
	const double2 &loc = dataLocal.loc;
	int ret = wall.intersection2LineSeg(
		loc.x,
		loc.y,
		loc.x + 0.5 * newVelo.x * tick,
		loc.y + 0.5 * newVelo.y * tick,
		crx,
		cry
		);
	if (ret == 1)
	{
		if (fabs(crx - loc.x) > 0)
			tt = (crx - loc.x) / (newVelo.x * tick);
		else
			tt = (crx - loc.y) / (newVelo.y * tick + 1e-20);
		if (tt < mint)
			mint = tt;
	}
}
__device__ void SocialForceAgent::computeDirection(const SocialForceAgentData &dataLocal, double2 &dvt) {
	//my data
	const double2& loc = dataLocal.loc;
	const double2& goal = dataLocal.goal;
	const double2& velo = dataLocal.velocity;
	const double& v0 = dataLocal.v0;
	const double& mass = dataLocal.mass;

	dvt.x = 0;	dvt.y = 0;
	double2 diff; diff.x = 0; diff.y = 0;
	double d0 = sqrt((loc.x - goal.x) * (loc.x - goal.x) + (loc.y - goal.y) * (loc.y - goal.y));
	diff.x = v0 * (goal.x - loc.x) / d0;
	diff.y = v0 * (goal.y - loc.y) / d0;
	dvt.x = (diff.x - velo.x) / tao;
	dvt.y = (diff.y - velo.y) / tao;
}

__device__ int sharedMinAndMax(int value, bool minFlag) {
	for (int i = 16; i >= 1; i /= 2) {
		if (minFlag)
			value = min(value, __shfl_xor(value, i, 32));
		else
			value = max(value, __shfl_xor(value, i, 32));
	}
	return value;
}

__device__ void SocialForceAgent::computeSocialForceRoom(SocialForceAgentData &dataLocal, double2 &fSum) {
	//__shared__ SocialForceAgentData sdata[BLOCK_SIZE];
	fSum.x = 0; fSum.y = 0;
	double ds = 0;
	int neighborCount = 0;

	int wid = threadIdx.x >> 5;
	int lane = threadIdx.x & 31;

	int cidStart = 0;
	int cidEnd = NUM_CAP;

	//while (cidStart < cidEnd) {
		//if (cidStart + threadIdx.x < cidEnd) {
		//	SocialForceAgent *other = myClone->context[cidStart + threadIdx.x];
		//	sdata[threadIdx.x] = other->data;
		//}

		//int iterCount = cidEnd - cidStart > BLOCK_SIZE ? BLOCK_SIZE : cidEnd - cidStart;

	for (int i = 0; i < NUM_CAP; i++) {
		SocialForceAgent *other = myClone->context[i];
		SocialForceAgentData otherData = other->data;
		ds = length(otherData.loc - dataLocal.loc);
		if (ds < 6 && ds > 0) {
			neighborCount++;
			computeIndivSocialForceRoom(dataLocal, otherData, fSum);

			for (int i = 0; i < NUM_PARAM; i++)
				this->flagCloning[i] |= other->flagCloning[i];
		}
	}

	//	cidStart += BLOCK_SIZE;
	//}

	/*
	for (int i = 0; i < NUM_CAP; i++) {
		SocialForceAgent *other = myClone->context[i];
		SocialForceAgentData otherData = other->data;
		ds = length(otherData.loc - dataLocal.loc);
		if (ds < 6 && ds > 0) {
			neighborCount++;
			computeIndivSocialForceRoom(dataLocal, otherData, fSum);
		}
	}
	*/

	dataLocal.numNeighbor = neighborCount;
}
__device__ void SocialForceAgent::chooseNewGoal(const double2 &newLoc, double epsilon, double2 &newGoal) {
	double2 oldGoal = newGoal;
	double2 center = make_double2(ENV_DIM / 2, ENV_DIM / 2);
	if (newLoc.x < center.x && newLoc.y <= center.y) {
		newGoal.x = 0.5 * ENV_DIM;
		newGoal.y = 0.3 * ENV_DIM;
	}
	else if (newLoc.x <= center.x && newLoc.y > center.y) {
		newGoal.x = 0.3 * ENV_DIM;
		newGoal.y = 0.5 * ENV_DIM;
	}
	else if (newLoc.x > center.x && newLoc.y > center.y) {
		newGoal.x = 0.5 * ENV_DIM;
		newGoal.y = 0.7 * ENV_DIM;
	}
	else if (newLoc.x >= center.x && newLoc.y < center.y){
		newGoal.x = 0.9 * ENV_DIM;
		newGoal.y = 0.3 * ENV_DIM;
	}
}
__device__ void SocialForceAgent::step(){

	double cMass = 100;

	const double2& loc = data.loc;
	const double2& goal = data.goal;
	const double2& velo = data.velocity;
	const double& v0 = data.v0;
	const double& mass = data.mass;

	//compute the direction
	double2 dvt;
	computeDirection(data, dvt);

	//compute force with other agents
	double2 fSum;
	computeSocialForceRoom(data, fSum);

	//compute force with walls and gates
	for (int i = 0; i < NUM_WALLS; i++) {
		obstacleLine wall = myClone->walls[i];
		computeForceWithWall(data, wall, cMass, fSum);
	}

	for (int i = 0; i < NUM_PARAM; i++) {
		obstacleLine &gate = myClone->gates[i];
		if (gate.pointToLineDist(loc) < 6) {
			// ideally, parent clone agent should compare against all child clone parameter configuration
			this->flagCloning[i] = -1;
		}
	}

	//sum up
	dvt.x += fSum.x / mass;
	dvt.y += fSum.y / mass;

	double2 newVelo = data.velocity;
	double2 newLoc = data.loc;
	double2 newGoal = data.goal;

	double tick = 0.1;
	newVelo.x += dvt.x * tick * (1);// + this->random->gaussian() * 0.1);
	newVelo.y += dvt.y * tick * (1);// + this->random->gaussian() * 0.1);
	double dv = sqrt(newVelo.x * newVelo.x + newVelo.y * newVelo.y);

	if (dv > maxv) {
		newVelo.x = newVelo.x * maxv / dv;
		newVelo.y = newVelo.y * maxv / dv;
	}

	double mint = 1;
	for (int i = 0; i < NUM_WALLS; i++) {
		obstacleLine wall = myClone->walls[i];
		computeWallImpaction(data, wall, newVelo, tick, mint);
	}

	newVelo.x *= mint;
	newVelo.y *= mint;
	newLoc.x += newVelo.x * tick;
	newLoc.y += newVelo.y * tick;

	double goalTemp = goal.x;

	chooseNewGoal(newLoc, mass / cMass, newGoal);

	newLoc.x = correctCrossBoader(newLoc.x, ENV_DIM);
	newLoc.y = correctCrossBoader(newLoc.y, ENV_DIM);

	dataCopy = data;

	dataCopy.loc = newLoc;
	dataCopy.velocity = newVelo;
	dataCopy.goal = newGoal;
}
__device__ void SocialForceAgent::init(SocialForceClone* c, int idx) {
	this->contextId = idx;
	//this->myOrigin = NULL;
	this->goalIdx = 0;
	this->myClone = c;

	for (int i = 0; i < NUM_PARAM; i++) {
		this->flagCloning[i] = 0;
		this->flagCloned[i] = 0;
	}

	hiprandState_t rStateLocal = c->rState[idx];
	this->color.x = hiprand(&rStateLocal) % 256;
	this->color.y = hiprand(&rStateLocal) % 256;
	this->color.z = hiprand(&rStateLocal) % 256;
	this->color.w = hiprand(&rStateLocal) % 256;
	
	SocialForceAgentData & dataLocal = this->data; //= &sfModel->originalAgents->dataArray[dataSlot];
	float rx = (float)(idx / 32) / (float)32;
	float ry = (float)(idx % 32) / (float)32;
	dataLocal.loc.x = (0.6 + 0.1 * hiprand_uniform(&rStateLocal)) * ENV_DIM;
	dataLocal.loc.y = (0.5 + 0.4 * hiprand_uniform(&rStateLocal)) * ENV_DIM;

	dataLocal.velocity.x = 2;//4 * (this->random->uniform()-0.5);
	dataLocal.velocity.y = 2;//4 * (this->random->uniform()-0.5);

	dataLocal.v0 = 2;
	dataLocal.mass = 50;
	dataLocal.numNeighbor = 0;
	//chooseNewGoal(dataLocal.loc, 0, dataLocal.goal);

	dataLocal.goal = make_double2(0.5 * ENV_DIM, 0.7 * ENV_DIM);
	this->dataCopy = dataLocal;
}

__device__ void SocialForceAgent::initNewClone(SocialForceAgent *parent, SocialForceClone *childClone) {
	this->color = childClone->color;
	this->contextId = parent->contextId;
	//this->myOrigin = parent;
	this->myClone = childClone;
	this->goalIdx = parent->goalIdx;
	for (int i = 0; i < NUM_GOAL; i++)
		this->goalSeq[i] = parent->goalSeq[i];

	for (int i = 0; i < NUM_PARAM; i++) {
		this->flagCloning[i] = 0;
		this->flagCloned[i] = 0;
	}

	this->data = parent->data;
	this->dataCopy = parent->dataCopy;

}

namespace clone {
	__global__ void stepKernel(SocialForceClone *c, int numElemLocal) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < numElemLocal)
			c->ap->agentPtrArray[index]->step();
	}
	__global__ void swapKernel(SocialForceClone *c, int numElemLocal) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numElemLocal) {
			SocialForceAgent &agent = *c->ap->agentPtrArray[idx];
			agent.data = agent.dataCopy;
		}
	}
}

void SocialForceClone::step(int stepCount) {
	if (numElem == 0)
		return;
	int gSize;

	//alterGate(stepCount);

	/*
	hipMemcpyAsync(contextSorted, context, sizeof(SocialForceAgent*) * NUM_CAP, hipMemcpyDeviceToDevice, myStream);
	hipStreamSynchronize(myStream);
	NeighborModule::sortAgentByLocKernel << <1, 1, 0, myStream >> >(this->contextSorted, this->rState, NUM_CAP);
	hipMemsetAsync(cidStarts, 0xff, sizeof(int) * NUM_CELL * NUM_CELL, myStream);
	hipMemsetAsync(cidEnds, 0xff, sizeof(int) * NUM_CELL * NUM_CELL, myStream);
	hipStreamSynchronize(myStream);
	gSize = GRID_SIZE(NUM_CAP);
	NeighborModule::setCidStartEndKernel<<<gSize, BLOCK_SIZE, 0, myStream>>>(contextSorted, cidStarts, cidEnds, NUM_CAP);
	NeighborModule::sortAgentByLocKernel << <1, 1, 0, myStream >> >(this->apHost->agentPtrArray, this->rState, this->numElem);
	*/
	
	gSize = GRID_SIZE(numElem);
	size_t smemSize = sizeof(SocialForceAgentData) * BLOCK_SIZE;
	clone::stepKernel << <gSize, BLOCK_SIZE, smemSize, myStream >> >(selfDev, numElem);
	//clone::stepKernel << <gSize, BLOCK_SIZE >> >(selfDev, numElem);
}

void SocialForceClone::swap() {
	if (numElem == 0)
		return;
	int gSize = GRID_SIZE(numElem);
	clone::swapKernel << <gSize, BLOCK_SIZE >> >(selfDev, numElem);
}

void SocialForceClone::alterGate(int stepCount) {
	bool changed = false;
	for (int i = 0; i < NUM_PARAM; i++) {
		if (cloneParams[i] == stepCount) {
			changed = true;
			gates[i].init(0, 0, 0, 0);
			//hipMemcpyAsync(&selfDev->gates[i], &gates[i], sizeof(obstacleLine), hipMemcpyHostToDevice, myStream);
			hipMemcpy(&selfDev->gates[i], &gates[i], sizeof(obstacleLine), hipMemcpyHostToDevice);
		}
	}
}

namespace AppUtil {

	__device__ bool cloningCondition(SocialForceAgent *agent,
		SocialForceClone *parentClone, SocialForceClone *childClone) {

		// if agent has been cloned?
		if (childClone->cloneFlags[agent->contextId] == true)
			return false;

		// active cloning condition
		double2 &loc = agent->data.loc;
		for (int i = 0; i < NUM_PARAM; i++) {
			int param1 = parentClone->cloneParams[i];
			int param2 = childClone->cloneParams[i];
			if (param1 != param2) {
				obstacleLine g1 = parentClone->gates[i];
				obstacleLine g2 = childClone->gates[i];
				if (g1.pointToLineDist(loc) < 6)
					return true;
				if (g2.pointToLineDist(loc) < 6)
					return true;
			}
		}

		// passive cloning condition
#define MY_MAX(a, b) (a > b ? a : b)
#define MY_MIN(a, b) (a < b ? a : b)
		int minx = MY_MAX((loc.x - RADIUS_I) / CELL_DIM, 0);
		int miny = MY_MAX((loc.y - RADIUS_I) / CELL_DIM, 0);
		int maxx = MY_MIN((loc.x + RADIUS_I) / CELL_DIM, NUM_CELL - 1);
		int maxy = MY_MIN((loc.y + RADIUS_I) / CELL_DIM, NUM_CELL - 1);
		for (int i = minx; i <= maxx; i++)
			for (int j = miny; j <= maxy; j++)
				if (childClone->takenMap[i * NUM_CELL + j])
					return true;

		// pass all the check, don't need to be cloned
		return false;
	}

	__global__ void updateContextKernel(SocialForceClone *c, int numElem) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numElem) {
			SocialForceAgent *agent = c->ap->agentPtrArray[idx];
			c->context[agent->contextId] = agent;
		}
	}

	__global__ void constructPassiveMap(SocialForceClone *c, int numElem) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numElem) {
			SocialForceAgent &agent = *c->ap->agentPtrArray[idx];
			int takenId = agent.data.loc.x / CELL_DIM;
			takenId = takenId * NUM_CELL + agent.data.loc.y / CELL_DIM;
			c->takenMap[takenId] = true;
		}
	}

	__global__ void performCloningKernel(SocialForceClone *p, SocialForceClone *c, int numCap) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numCap) {
			SocialForceAgent *agent = p->context[idx];
			if (cloningCondition(agent, p, c)) {
				uint lastNum = atomicInc(&c->numElem, numCap);
				SocialForceAgent& childAgent = *c->ap->agentPtrArray[lastNum];
				c->ap->takenFlags[lastNum] = true;
				childAgent.initNewClone(agent, c);
				c->context[childAgent.contextId] = &childAgent;
				c->cloneFlags[childAgent.contextId] = true;
				//c->numElem++; /* not written back */
			}
		}
	}

	__global__ void performCloningKernel(SocialForceClone *p, SocialForceClone *c, int numCap) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numCap) {

		}
	}
	
	__global__ void compareAndEliminateKernel(SocialForceClone *p, SocialForceClone *c, int numElem) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < numElem) {
			SocialForceAgent &childAgent = *c->ap->agentPtrArray[idx];
			SocialForceAgent &parentAgent = *p->context[childAgent.contextId]; // *(SocialForceAgent*)childAgent.myOrigin;
			double velDiff = length(childAgent.dataCopy.velocity - parentAgent.dataCopy.velocity);
			double locDiff = length(childAgent.dataCopy.loc - parentAgent.dataCopy.loc);
			if (locDiff == 0 && velDiff	== 0) {
				c->ap->takenFlags[idx] = false;
				c->cloneFlags[childAgent.contextId] = false;
			}
		}
	}

	template<class T>
	__device__  void swap(T * ar, int a, int b) {
		T t1 = ar[a];
		ar[a] = ar[b];
		ar[b] = t1;
	}

	__global__ void reorderKernel(SocialForceClone *c, int numElem) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx == 0) {
			int l = 0; int r = numElem;
			int i = l, j = l;
			for (; j < r; j++) {
				if (c->ap->takenFlags[j] == true) {
					swap<SocialForceAgent*>(c->ap->agentPtrArray, i, j);
					swap<bool>(c->ap->takenFlags, i, j);
					i++;
				}
			}
			c->numElem = i;
		}
	}

};

void SocialForceSimApp::performClone(SocialForceClone *parentClone, SocialForceClone *childClone) {
	childClone->parentCloneid = parentClone->cloneid;

	// 1. copy the context of parent clone
	hipMemcpyAsync(childClone->context, parentClone->context, NUM_CAP * sizeof(SocialForceAgent*), hipMemcpyDeviceToDevice, childClone->myStream);
	hipStreamSynchronize(childClone->myStream);
	//hipMemcpy(childClone->context, parentClone->context, NUM_CAP * sizeof(SocialForceAgent*), hipMemcpyDeviceToDevice);
	getLastCudaError("perform clone");

	// 2. update the context with agents of its own
	if (childClone->numElem > 0) {
		int gSize = GRID_SIZE(childClone->numElem);
		AppUtil::updateContextKernel << <gSize, BLOCK_SIZE, 0, childClone->myStream >> >(childClone->selfDev, childClone->numElem);
		//AppUtil::updateContextKernel << <gSize, BLOCK_SIZE >> >(childClone->selfDev, childClone->numElem);
	}
	getLastCudaError("perform clone");

	// 3. construct passive cloning map
	if (childClone->numElem > 0) {
		hipMemsetAsync(childClone->selfDev->takenMap, 0, sizeof(bool) * NUM_CELL * NUM_CELL, childClone->myStream);
		hipStreamSynchronize(childClone->myStream);
		//hipMemset(childClone->selfDev->takenMap, 0, sizeof(bool) * NUM_CELL * NUM_CELL);
		int gSize = GRID_SIZE(childClone->numElem);
		AppUtil::constructPassiveMap << <gSize, BLOCK_SIZE, 0, childClone->myStream >> >(childClone->selfDev, childClone->numElem);
		//AppUtil::constructPassiveMap << <gSize, BLOCK_SIZE >> >(childClone->selfDev, childClone->numElem);
	}
	getLastCudaError("perform clone");

	// 4. perform active and passive cloning (in cloningCondition checking)
	int gSize = GRID_SIZE(NUM_CAP);
	//AppUtil::performCloningKernel << <gSize, BLOCK_SIZE >> >(parentClone->selfDev, childClone->selfDev, NUM_CAP);
	AppUtil::performCloningKernel << <gSize, BLOCK_SIZE, 0, childClone->myStream >> >(parentClone->selfDev, childClone->selfDev, NUM_CAP);
	hipMemcpyAsync(childClone, childClone->selfDev, sizeof(SocialForceClone), hipMemcpyDeviceToHost, childClone->myStream);
	hipStreamSynchronize(childClone->myStream);
	getLastCudaError("perform clone");

}

void compareAndEliminateCPU(SocialForceClone *parentClone, SocialForceClone *childClone)
{
	wchar_t message[20];
	for (int i = 0; i < childClone->numElem; i++) {
		SocialForceAgent &childAgent = *childClone->ap->agentPtrArray[i];
		SocialForceAgent parentAgent; // *(SocialForceAgent*)childAgent.myOrigin;
		if (length(childAgent.dataCopy.velocity - parentAgent.dataCopy.velocity) == 0 &&
			length(childAgent.dataCopy.loc - parentAgent.dataCopy.loc) == 0) {
			childClone->ap->takenFlags[i] = false;
			childClone->cloneFlags[childAgent.contextId] = false;
		}
		/*else {
		if (childClone->cloneid == 4) {
		swprintf_s(message, 20, L"not false: %d\n", i);
		OutputDebugString(message);
		}
		}*/
	}
	childClone->numElem = childClone->ap->reorder(childClone->numElem);
}

void SocialForceSimApp::compareAndEliminate(SocialForceClone *parentClone, SocialForceClone *childClone) {
	if (childClone->numElem == 0) return;
	int gSize = GRID_SIZE(childClone->numElem);
	AppUtil::compareAndEliminateKernel << <gSize, BLOCK_SIZE, 0, childClone->myStream >> >(parentClone->selfDev, childClone->selfDev, childClone->numElem);
	//AppUtil::compareAndEliminateKernel << <gSize, BLOCK_SIZE>> >(parentClone->selfDev, childClone->selfDev, childClone->numElem);
	gSize = GRID_SIZE(NUM_CAP);
	AppUtil::reorderKernel << <1, 1, 0, childClone->myStream >> >(childClone->selfDev, childClone->numElem);
	//AppUtil::reorderKernel << <1, 1 >> >(childClone->selfDev, childClone->numElem);
	hipMemcpyAsync(childClone, childClone->selfDev, sizeof(SocialForceClone), hipMemcpyDeviceToHost, childClone->myStream);
	hipStreamSynchronize(childClone->myStream);
}

void SocialForceSimApp::proc(int p, int c, bool o, char *s) {
	performClone(cAll[p], cAll[c]);
	cAll[c]->step(stepCount);
	if (o) {
		if (stepCount < 800)
			cAll[c]->output(stepCount, s);
	}
	compareAndEliminate(cAll[p], cAll[c]);
}

void swap(int **cloneTree, int a, int b) {
	int t1 = cloneTree[0][a];
	cloneTree[0][a] = cloneTree[0][b];
	cloneTree[0][b] = t1;

	t1 = cloneTree[1][a];
	cloneTree[1][a] = cloneTree[1][b];
	cloneTree[1][b] = t1;
}

void quickSort(int **cloneTree, int l, int r) {
	if (l == r)
		return;
	int pi = l + rand() % (r - l);
	swap(cloneTree, l, pi);
	int pivot = cloneTree[0][l];

	int i = l + 1, j = l + 1;
	for (; j < r; j++) {
		if (cloneTree[0][j] < pivot) {
			swap(cloneTree, i, j);
			i++;
		}
	}
	swap(cloneTree, l, i - 1);
	quickSort(cloneTree, l, i - 1);
	quickSort(cloneTree, i, r);
}

void SocialForceSimApp::mst() {
	// clone diff matrix
	int **cloneDiff = new int*[totalClone];
	for (int i = 0; i < totalClone; i++) {
		cloneDiff[i] = new int[totalClone];
		for (int j = 0; j < totalClone; j++)
			cloneDiff[i][j] = 0;
	}

	for (int i = 0; i < totalClone; i++) {
		for (int j = 0; j < totalClone; j++) {
			for (int k = 0; k < NUM_PARAM; k++) {
				if (cAll[i]->cloneParams[k] != cAll[j]->cloneParams[k])
					cloneDiff[i][j]++;
			}
			wchar_t message[20];
			swprintf_s(message, 20, L"%d ", cloneDiff[i][j]);
			OutputDebugString(message);
		}
		OutputDebugString(L"\n");
	}
	int *parent = cloneTree[0] = new int[totalClone];
	int *child = cloneTree[1] = new int[totalClone];
	int *key = new int[totalClone];
	bool *mstSet = new bool[totalClone];

	for (int i = 0; i < totalClone; i++)
		child[i] = i, key[i] = INT_MAX, mstSet[i] = false;

	key[0] = 0;
	parent[0] = -1;
	child[0] = 0;

	int count = 0;
	while (count++ < totalClone - 1) {
		int minKey = INT_MAX;
		int minIdx;
		for (int j = 0; j < totalClone; j++)
			if (mstSet[j] == false && key[j] < minKey)
				minKey = key[j], minIdx = j;
		mstSet[minIdx] = true;

		for (int j = 0; j < totalClone; j++)
			if (cloneDiff[minIdx][j] && mstSet[j] == false && cloneDiff[minIdx][j] < key[j])
				parent[j] = minIdx, key[j] = cloneDiff[minIdx][j];
	}

	quickSort(cloneTree, 0, totalClone);

	for (int i = 1; i < totalClone; i++) {
		wchar_t message[20];
		swprintf_s(message, 20, L"%d - %d: %d\n", cloneTree[0][i], cloneTree[1][i], cloneDiff[i][parent[i]]);
		OutputDebugString(message);
	}

	delete mstSet;
	delete key;
}

__global__ void getLocAndColorKernel(SocialForceClone *c, double2 *loc, uchar4 *color, int *contextId, int numElem) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numElem) {
		loc[idx] = c->context[idx]->data.loc;
		color[idx] = c->context[idx]->color;
		contextId[idx] = c->context[idx]->contextId;
	}
}

void SocialForceSimApp::getLocAndColorFromDevice(){
	SocialForceClone *c = cAll[paintId];
	int gSize = GRID_SIZE(NUM_CAP);
	getLocAndColorKernel << <gSize, BLOCK_SIZE >> >(c->selfDev, debugLocDev, debugColorDev, debugContextIdDev, NUM_CAP);
	hipMemcpy(debugLocHost, debugLocDev, sizeof(double2) * NUM_CAP, hipMemcpyDeviceToHost);
	hipMemcpy(debugColorHost, debugColorDev, sizeof(uchar4) * NUM_CAP, hipMemcpyDeviceToHost);
	hipMemcpy(debugContextIdHost, debugContextIdDev, sizeof(int) * NUM_CAP, hipMemcpyDeviceToHost);
	hipMemcpy(c->takenMap, c->selfDev->takenMap, sizeof(bool) * NUM_CELL * NUM_CELL, hipMemcpyDeviceToHost);
	//hipMemcpy(debugCidStartsHost, c->cidStarts, sizeof(int) * NUM_CELL * NUM_CELL, hipMemcpyDeviceToHost);
	//hipMemcpy(debugCidEndsHost, c->cidEnds, sizeof(int) * NUM_CELL * NUM_CELL, hipMemcpyDeviceToHost);
	//wchar_t message[128];
	//for (int i = 0; i < NUM_CELL * NUM_CELL; i++) {
	//	swprintf_s(message, L"(%d, %d) ", debugCidStartsHost[i], debugCidEndsHost[i]);
	//	OutputDebugString(message);
	//}
	//OutputDebugString(L"\n");
}

__global__ void initRandomKernel(SocialForceClone* c, int numElemLocal) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numElemLocal) {
		hiprand_init(1234, idx, 0, &c->rState[idx]);
	}
}

__global__ void initRootCloneKernel(SocialForceClone* c, int numElemLocal) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numElemLocal) {
		c->ap->agentArray[idx].init(c, idx);
		c->context[idx] = &c->ap->agentArray[idx];
		c->cloneFlags[idx] = false;
	}
	if (idx == 0)
		c->numElem = numElemLocal;
}

void SocialForceSimApp::initRootClone(SocialForceClone* cHost, SocialForceClone* cDev) {
	cHost->numElem = NUM_CAP;

	int gSize = GRID_SIZE(NUM_CAP);
	initRandomKernel << <gSize, BLOCK_SIZE >> >(cDev, NUM_CAP);
	initRootCloneKernel << <gSize, BLOCK_SIZE >> >(cDev, NUM_CAP);
}