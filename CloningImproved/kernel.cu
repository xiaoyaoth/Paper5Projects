#include "gsimcore.cuh"
//#include "boid.cuh"
#ifdef _WIN32
#include <Windows.h>
#else
#include <sys/time.h>
#endif
//#include "test.cuh"
//#include "socialForce.cuh"
#include "socialForceEnhanced.cuh"
int main(int argc, char *argv[]){
	//argv[1]: config.txt
	//argv[2]: numAgent
	init<SocialForceRoomAgentData>(argv[1]);
	SocialForceRoomModel *model_h = new SocialForceRoomModel(&argv[2]);
	/*Main work started here*/

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	doLoop(model_h);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("execution time: %f\n", time);
}
